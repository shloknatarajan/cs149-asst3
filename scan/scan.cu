#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

#include "CycleTimer.h"

#define THREADS_PER_BLOCK 256


// helper function to round an integer up to the next power of 2
static inline int nextPow2(int n) {
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}

__global__ void mark_repeats_kernel(int* input, int length, int* flags) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < length - 1) {
        flags[idx] = (input[idx] == input[idx + 1]) ? 1 : 0;
    } else if (idx == length - 1) {
        flags[idx] = 0; 
    }
}

__global__ void scatter_indices_kernel(int* flags, int* scanned, int length, int* output) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < length - 1 && flags[idx] == 1) {
        output[scanned[idx]] = idx;
    }
}

__global__ void upsweep_kernel(int* output, int num_iterations, int two_d, int two_dplus1) {
    int iter = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (iter < num_iterations) {
        int i = iter * two_dplus1;
        output[i + two_dplus1 - 1] += output[i + two_d - 1];
    }
}

__global__ void downsweep_kernel(int* output, int num_iterations, int two_d, int two_dplus1) {
    int iter = blockIdx.x * blockDim.x + threadIdx.x;
  
    if (iter < num_iterations) {
        int i = iter * two_dplus1;
        int t = output[i + two_d - 1];
        output[i + two_d - 1] = output[i + two_dplus1 - 1];
        output[i + two_dplus1 - 1] += t;
    }
}

void exclusive_scan(int* input, int N, int* result)
{
    const int threadsPerBlock = 256;
    const int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;
    // CS149 TODO:
    //
    // Implement your exclusive scan implementation here.  Keep in
    // mind that although the arguments to this function are device
    // allocated arrays, this is a function that is running in a thread
    // on the CPU.  Your implementation will need to make multiple calls
    // to CUDA kernel functions (that you must write) to implement the
    // scan.
    int rounded_N = nextPow2(N);
    
    // Upsweep
    for (int two_d = 1; two_d <= rounded_N/2; two_d *= 2) {
        int two_dplus1 = 2 * two_d;
        int num_iterations = rounded_N / two_dplus1;
        
        const int threadsPerBlock = 256;
        const int blocks = (num_iterations + threadsPerBlock - 1) / threadsPerBlock;
        
        upsweep_kernel<<<blocks, threadsPerBlock>>>(result, num_iterations, two_d, two_dplus1);
    }

    // Set last to 0
    hipMemset(&result[rounded_N - 1], 0, sizeof(int));
    
    // Downsweep
    for (int two_d = rounded_N/2; two_d >= 1; two_d /= 2) {
        int two_dplus1 = 2 * two_d;
        int num_iterations = rounded_N / two_dplus1;
        
        const int threadsPerBlock = 256;
        const int blocks = (num_iterations + threadsPerBlock - 1) / threadsPerBlock;
        
        downsweep_kernel<<<blocks, threadsPerBlock>>>(result, num_iterations, two_d, two_dplus1);
    }
}
int* do_scatter(int* flags, int* scanned, int length, int* output) {
    int threadsPerBlock = 256;
    int blocks = (length + threadsPerBlock - 1) / threadsPerBlock;
    scatter_indices_kernel<<<blocks, threadsPerBlock>>>(flags, scanned, length, output);
    return output;
}
int* getRepeatsFlags(int* input, int length) {
    int threadsPerBlock = 256;
    int blocks = (length + threadsPerBlock - 1) / threadsPerBlock;
    int* device_flags;
    hipMalloc((void **)&device_flags, sizeof(int) * length);
    mark_repeats_kernel<<<blocks, threadsPerBlock>>>(input, length, device_flags);
    return device_flags;
}
//
// cudaScan --
//
// This function is a timing wrapper around the student's
// implementation of scan - it copies the input to the GPU
// and times the invocation of the exclusive_scan() function
// above. Students should not modify it.
double cudaScan(int* inarray, int* end, int* resultarray)
{
    int* device_result;
    int* device_input;
    int N = end - inarray;  

    // This code rounds the arrays provided to exclusive_scan up
    // to a power of 2, but elements after the end of the original
    // input are left uninitialized and not checked for correctness.
    //
    // Student implementations of exclusive_scan may assume an array's
    // allocated length is a power of 2 for simplicity. This will
    // result in extra work on non-power-of-2 inputs, but it's worth
    // the simplicity of a power of two only solution.

    int rounded_length = nextPow2(end - inarray);
    
    hipMalloc((void **)&device_result, sizeof(int) * rounded_length);
    hipMalloc((void **)&device_input, sizeof(int) * rounded_length);

    // For convenience, both the input and output vectors on the
    // device are initialized to the input values. This means that
    // students are free to implement an in-place scan on the result
    // vector if desired.  If you do this, you will need to keep this
    // in mind when calling exclusive_scan from find_repeats.
    hipMemcpy(device_input, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_result, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    exclusive_scan(device_input, N, device_result);

    // Wait for completion
    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
       
    hipMemcpy(resultarray, device_result, (end - inarray) * sizeof(int), hipMemcpyDeviceToHost);

    double overallDuration = endTime - startTime;
    return overallDuration; 
}


// cudaScanThrust --
//
// Wrapper around the Thrust library's exclusive scan function
// As above in cudaScan(), this function copies the input to the GPU
// and times only the execution of the scan itself.
//
// Students are not expected to produce implementations that achieve
// performance that is competition to the Thrust version, but it is fun to try.
double cudaScanThrust(int* inarray, int* end, int* resultarray) {

    int length = end - inarray;
    thrust::device_ptr<int> d_input = thrust::device_malloc<int>(length);
    thrust::device_ptr<int> d_output = thrust::device_malloc<int>(length);
    
    hipMemcpy(d_input.get(), inarray, length * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    thrust::exclusive_scan(d_input, d_input + length, d_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
   
    hipMemcpy(resultarray, d_output.get(), length * sizeof(int), hipMemcpyDeviceToHost);

    thrust::device_free(d_input);
    thrust::device_free(d_output);

    double overallDuration = endTime - startTime;
    return overallDuration; 
}


// find_repeats --
//
// Given an array of integers `device_input`, returns an array of all
// indices `i` for which `device_input[i] == device_input[i+1]`.
//
// Returns the total number of pairs found
int find_repeats(int* device_input, int length, int* device_output) {

    // CS149 TODO:
    //
    // Implement this function. You will probably want to
    // make use of one or more calls to exclusive_scan(), as well as
    // additional CUDA kernel launches.
    //    
    // Note: As in the scan code, the calling code ensures that
    // allocated arrays are a power of 2 in size, so you can use your
    // exclusive_scan function with them. However, your implementation
    // must ensure that the results of find_repeats are correct given
    // the actual array length.
    int rounded_length = nextPow2(length);
    int* device_scanned;
    hipMalloc((void **)&device_scanned, rounded_length * sizeof(int));
    hipMemset(device_scanned, 0, rounded_length * sizeof(int));
    
    int* device_flags = getRepeatsFlags(device_input, length);
    
    int last_flag;
    hipMemcpy(&last_flag, &device_flags[length - 1], sizeof(int), hipMemcpyDeviceToHost);
    printf("last_flag: %d\n", last_flag);
    fflush(stdout);

    hipMemcpy(device_scanned, device_flags, length * sizeof(int), hipMemcpyDeviceToDevice);
    exclusive_scan(device_flags, rounded_length, device_scanned);
    
    int last_scan_value;
    hipMemcpy(&last_scan_value, &device_scanned[length - 1], sizeof(int), hipMemcpyDeviceToHost);
    int num_total_flags = last_scan_value + last_flag;
    
    do_scatter(device_flags, device_scanned, length, device_output);
    
    hipFree(device_flags);
    hipFree(device_scanned);
    return num_total_flags; 
}


//
// cudaFindRepeats --
//
// Timing wrapper around find_repeats. You should not modify this function.
double cudaFindRepeats(int *input, int length, int *output, int *output_length) {

    int *device_input;
    int *device_output;
    int rounded_length = nextPow2(length);
    
    hipMalloc((void **)&device_input, rounded_length * sizeof(int));
    hipMalloc((void **)&device_output, rounded_length * sizeof(int));
    hipMemcpy(device_input, input, length * sizeof(int), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    double startTime = CycleTimer::currentSeconds();
    
    int result = find_repeats(device_input, length, device_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    // set output count and results array
    *output_length = result;
    hipMemcpy(output, device_output, length * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(device_input);
    hipFree(device_output);

    float duration = endTime - startTime; 
    return duration;
}



void printCudaInfo()
{
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n"); 
}
